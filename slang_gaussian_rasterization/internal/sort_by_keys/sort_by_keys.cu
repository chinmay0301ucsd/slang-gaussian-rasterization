#include <torch/extension.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <vector>

namespace extension_cpp {

  std::tuple<torch::Tensor, torch::Tensor>
  sort_by_keys(
    const at::Tensor keys,
    const at::Tensor values,
    const int highest_tile_id_msb)
  {
    TORCH_CHECK(keys.sizes() == values.sizes());
    TORCH_CHECK(keys.dtype() == torch::kLong);
    TORCH_CHECK(values.dtype() == torch::kInt32);
    TORCH_INTERNAL_ASSERT(keys.device().type() == at::DeviceType::CUDA);
    TORCH_INTERNAL_ASSERT(values.device().type() == at::DeviceType::CUDA);

    at::Tensor keys_sorted = torch::empty(keys.sizes(), keys.options());
    at::Tensor values_sorted = torch::empty(values.sizes(), values.options());

    at::Tensor keys_contig = keys.contiguous();
    at::Tensor values_contig = values.contiguous();
    at::Tensor keys_sorted_contig = keys_sorted.contiguous();
    at::Tensor values_sorted_contig = values_sorted.contiguous();

    const int64_t* keys_ptr = keys_contig.data_ptr<int64_t>();
    const int32_t* values_ptr = values_contig.data_ptr<int32_t>();
    int64_t* keys_sorted_ptr = keys_sorted_contig.data_ptr<int64_t>();
    int32_t* values_sorted_ptr = values_sorted_contig.data_ptr<int32_t>();

    void     *d_temp_storage = nullptr;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(
      d_temp_storage, temp_storage_bytes,
      keys_ptr, keys_sorted_ptr,
      values_ptr, values_sorted_ptr,
      keys.sizes()[0]);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceRadixSort::SortPairs(
      d_temp_storage, temp_storage_bytes,
      keys_ptr, keys_sorted_ptr,
      values_ptr, values_sorted_ptr,
      keys.sizes()[0], 0, 32 + highest_tile_id_msb);

    hipFree(d_temp_storage);

    return std::make_tuple(keys_sorted_contig, values_sorted_contig);
  }

  PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("sort_by_keys", &sort_by_keys);
  }

}
